#include "hip/hip_runtime.h"
#include "utils/metrics_utils.h"
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <stdexcept>
#include <string>

namespace MetricsUtils {

struct CompareLabels {
    __host__ __device__
    int operator()(const thrust::tuple<const int&, const int&>& t) const {
        return thrust::get<0>(t) == thrust::get<1>(t) ? 1 : 0;
    }
};

// Calculate accuracy by comparing predictions with true labels
float calculateAccuracy(const int* predictions, const int* labels, int n_samples) {
    try {
        thrust::device_ptr<const int> d_pred(predictions);
        thrust::device_ptr<const int> d_labels(labels);
        
        int correct = thrust::transform_reduce(
            thrust::make_zip_iterator(thrust::make_tuple(d_pred, d_labels)),
            thrust::make_zip_iterator(thrust::make_tuple(d_pred + n_samples, d_labels + n_samples)),
            CompareLabels(),
            0,
            thrust::plus<int>()
        );
        
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        
        return static_cast<float>(correct) / n_samples;
    } catch (const std::runtime_error& e) {
        throw std::runtime_error("Accuracy calculation failed: " + std::string(e.what()));
    }
}

// CUDA kernel for computing confusion matrix
__global__ void confusionMatrixKernel(const int* predictions,
                                     const int* labels,
                                     int* confusion_matrix,
                                     int n_samples,
                                     int n_classes) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < n_samples) {
        int pred = predictions[idx];
        int true_label = labels[idx];
        // Atomic add to safely update confusion matrix in parallel
        atomicAdd(&confusion_matrix[true_label * n_classes + pred], 1);
    }
}

// Calculate confusion matrix for multi-class classification
void calculateConfusionMatrix(const int* predictions,
                            const int* labels,
                            int n_samples,
                            int n_classes,
                            int* confusion_matrix) {
    // Allocate and initialize device memory for confusion matrix
    int* d_confusion_matrix;
    hipMalloc(&d_confusion_matrix, n_classes * n_classes * sizeof(int));
    hipMemset(d_confusion_matrix, 0, n_classes * n_classes * sizeof(int));
    
    // Configure kernel execution parameters
    dim3 block_size(BLOCK_SIZE);
    dim3 grid_size((n_samples + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    // Launch confusion matrix kernel
    confusionMatrixKernel<<<grid_size, block_size>>>(
        predictions,
        labels,
        d_confusion_matrix,
        n_samples,
        n_classes
    );
    
    // Copy results back to host and cleanup
    hipMemcpy(confusion_matrix, 
               d_confusion_matrix, 
               n_classes * n_classes * sizeof(int), 
               hipMemcpyDeviceToHost);
    
    hipFree(d_confusion_matrix);
}

// Calculate precision for a specific class
float calculatePrecision(const int* confusion_matrix, 
                        int class_idx, 
                        int n_classes) {
    // True positives are on the diagonal
    int true_positives = confusion_matrix[class_idx * n_classes + class_idx];
    int predicted_positives = 0;
    
    // Sum all predictions for this class (column sum)
    for (int i = 0; i < n_classes; i++) {
        predicted_positives += confusion_matrix[i * n_classes + class_idx];
    }
    
    // Return precision, handling division by zero
    return predicted_positives > 0 ? 
           static_cast<float>(true_positives) / predicted_positives : 0.0f;
}

// Calculate recall for a specific class
float calculateRecall(const int* confusion_matrix, 
                     int class_idx, 
                     int n_classes) {
    // True positives are on the diagonal
    int true_positives = confusion_matrix[class_idx * n_classes + class_idx];
    int actual_positives = 0;
    
    // Sum all actual instances of this class (row sum)
    for (int i = 0; i < n_classes; i++) {
        actual_positives += confusion_matrix[class_idx * n_classes + i];
    }
    
    // Return recall, handling division by zero
    return actual_positives > 0 ? 
           static_cast<float>(true_positives) / actual_positives : 0.0f;
}

// Calculate F1 score from precision and recall
float calculateF1Score(float precision, float recall) {
    // Return F1 score, handling division by zero
    return (precision + recall > 0.0f) ? 
           2.0f * precision * recall / (precision + recall) : 0.0f;
}

}
