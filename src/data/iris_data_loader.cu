#include "data/iris_data_loader.h"
#include <fstream>
#include <sstream>

IrisDataLoader::IrisDataLoader() {
    data.features = nullptr;
    data.labels = nullptr;
    data.n_samples = 0;
    data.n_features = 4;
    data.n_classes = 3;
}

IrisDataLoader::~IrisDataLoader() {
    freeMemory();
}

bool IrisDataLoader::loadData(IrisData& data) {
    std::vector<float> features;
    std::vector<int> labels;
    
    if (!loadFromFile(features, labels)) {
        return false;
    }
    
    int n_samples = labels.size();
    data.n_samples = n_samples;
    
    if (!allocateMemory(data, n_samples)) {
        return false;
    }
    
    // Copy data to device
    hipError_t error;
    error = hipMemcpy(data.features, features.data(), 
                      features.size() * sizeof(float), 
                      hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        freeMemory(data);
        return false;
    }
    
    error = hipMemcpy(data.labels, labels.data(), 
                      labels.size() * sizeof(int), 
                      hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        freeMemory(data);
        return false;
    }
    
    return true;
}

bool IrisDataLoader::allocateMemory(IrisData& data, int n_samples) {
    hipError_t error;
    
    error = hipMalloc(&data.features, n_samples * 4 * sizeof(float));
    if (error != hipSuccess) {
        return false;
    }
    
    error = hipMalloc(&data.labels, n_samples * sizeof(int));
    if (error != hipSuccess) {
        if (data.features) hipFree(data.features);
        return false;
    }
    
    return true;
}

void IrisDataLoader::freeMemory() {
    if (data.features) CUDA_CHECK(hipFree(data.features));
    if (data.labels) CUDA_CHECK(hipFree(data.labels));
}
