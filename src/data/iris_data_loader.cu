#include "data/iris_data_loader.h"
#include <fstream>
#include <sstream>

IrisDataLoader::IrisDataLoader() {
    data.features = nullptr;
    data.labels = nullptr;
    data.n_samples = 0;
    data.n_features = 4;
    data.n_classes = 3;
}

IrisDataLoader::~IrisDataLoader() {
    freeMemory();
}

bool IrisDataLoader::loadData(IrisData& data) {
    std::vector<float> features;
    std::vector<int> labels;
    
    if (!loadFromFile(features, labels)) {
        return false;
    }
    
    data.n_samples = labels.size();
    data.n_features = 4;
    data.n_classes = 3;
    
    // Allocate device memory
    if (hipMalloc(&data.features, features.size() * sizeof(float)) != hipSuccess) {
        return false;
    }
    if (hipMalloc(&data.labels, labels.size() * sizeof(int)) != hipSuccess) {
        hipFree(data.features);
        return false;
    }
    
    // Copy data to device
    if (hipMemcpy(data.features, features.data(), 
                   features.size() * sizeof(float), 
                   hipMemcpyHostToDevice) != hipSuccess) {
        hipFree(data.features);
        hipFree(data.labels);
        return false;
    }
    
    if (hipMemcpy(data.labels, labels.data(), 
                   labels.size() * sizeof(int), 
                   hipMemcpyHostToDevice) != hipSuccess) {
        hipFree(data.features);
        hipFree(data.labels);
        return false;
    }
    
    return true;
}

void IrisDataLoader::allocateMemory(int n_samples) {
    CUDA_CHECK(hipMalloc(&data.features, n_samples * 4 * sizeof(float)));
    CUDA_CHECK(hipMalloc(&data.labels, n_samples * sizeof(int)));
}

void IrisDataLoader::freeMemory() {
    if (data.features) CUDA_CHECK(hipFree(data.features));
    if (data.labels) CUDA_CHECK(hipFree(data.labels));
}
