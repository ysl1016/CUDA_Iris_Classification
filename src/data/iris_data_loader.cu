#include "data/iris_data_loader.h"
#include <fstream>
#include <sstream>

bool IrisDataLoader::loadData(IrisData& data) {
    std::vector<float> features;
    std::vector<int> labels;
    
    if (!loadFromFile(features, labels)) {
        return false;
    }
    
    int n_samples = labels.size();
    data.n_samples = n_samples;
    
    if (!allocateMemory(data, n_samples)) {
        return false;
    }
    
    // Copy data to device
    hipError_t error;
    error = hipMemcpy(data.features, features.data(), 
                      features.size() * sizeof(float), 
                      hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        freeMemory(data);
        return false;
    }
    
    error = hipMemcpy(data.labels, labels.data(), 
                      labels.size() * sizeof(int), 
                      hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        freeMemory(data);
        return false;
    }
    
    return true;
}

bool IrisDataLoader::allocateMemory(IrisData& data, int n_samples) {
    hipError_t error;
    
    error = hipMalloc(&data.features, n_samples * N_FEATURES * sizeof(float));
    if (error != hipSuccess) {
        return false;
    }
    
    error = hipMalloc(&data.labels, n_samples * sizeof(int));
    if (error != hipSuccess) {
        if (data.features) hipFree(data.features);
        return false;
    }
    
    return true;
}

void IrisDataLoader::freeMemory(IrisData& data) {
    if (data.features) {
        hipFree(data.features);
        data.features = nullptr;
    }
    if (data.labels) {
        hipFree(data.labels);
        data.labels = nullptr;
    }
}
