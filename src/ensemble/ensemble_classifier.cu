#include "hip/hip_runtime.h"
#include "ensemble/ensemble_classifier.h"
#include "utils/metrics_utils.h"
#include <thrust/execution_policy.h>

// Add constant for max epochs
#define MAX_EPOCHS 100

EnsembleClassifier::~EnsembleClassifier() {
    if (d_weights) CUDA_CHECK(hipFree(d_weights));
    if (d_predictions) CUDA_CHECK(hipFree(d_predictions));
}

__global__ void weightedVoteKernel(const int* individual_predictions,
                                 const float* weights,
                                 int* final_predictions,
                                 int n_samples,
                                 int n_classifiers,
                                 int n_classes) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < n_samples) {
        float* class_scores = new float[n_classes]();
        
        // Accumulate weighted votes
        for (int i = 0; i < n_classifiers; i++) {
            int pred = individual_predictions[idx * n_classifiers + i];
            class_scores[pred] += weights[i];
        }
        
        // Find class with maximum score
        int max_class = 0;
        float max_score = class_scores[0];
        for (int i = 1; i < n_classes; i++) {
            if (class_scores[i] > max_score) {
                max_score = class_scores[i];
                max_class = i;
            }
        }
        
        final_predictions[idx] = max_class;
        delete[] class_scores;
    }
}

void EnsembleClassifier::train(const IrisData& data) {
    // Train individual classifiers
    svm.train(data);
    nn.train(data, MAX_EPOCHS);
    kmeans.train(data);
    
    // Initialize weights equally
    float initial_weight = 1.0f / n_classifiers;
    thrust::fill(thrust::device, d_weights, d_weights + n_classifiers, initial_weight);
    
    // Update weights based on validation performance
    updateWeights(data.features, data.labels, data.n_samples);
}

void EnsembleClassifier::predict(const float* features, int* predictions, int n_samples) {
    // Get predictions from each classifier
    int* d_svm_pred;
    int* d_nn_pred;
    int* d_kmeans_pred;
    
    CUDA_CHECK(hipMalloc(&d_svm_pred, n_samples * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_nn_pred, n_samples * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_kmeans_pred, n_samples * sizeof(int)));
    
    svm.predict(features, d_svm_pred, n_samples);
    nn.predict(features, d_nn_pred, n_samples);
    kmeans.predict(features, d_kmeans_pred, n_samples);
    
    // Copy individual predictions to combined array
    CUDA_CHECK(hipMemcpy(d_predictions, d_svm_pred, n_samples * sizeof(int), hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipMemcpy(d_predictions + n_samples, d_nn_pred, n_samples * sizeof(int), hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipMemcpy(d_predictions + 2 * n_samples, d_kmeans_pred, n_samples * sizeof(int), hipMemcpyDeviceToDevice));
    
    // Combine predictions using weighted voting
    dim3 block_size(BLOCK_SIZE);
    dim3 grid_size((n_samples + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    weightedVoteKernel<<<grid_size, block_size>>>(
        d_predictions,
        d_weights,
        predictions,
        n_samples,
        n_classifiers,
        3  // n_classes for Iris dataset
    );
    
    // Cleanup
    CUDA_CHECK(hipFree(d_svm_pred));
    CUDA_CHECK(hipFree(d_nn_pred));
    CUDA_CHECK(hipFree(d_kmeans_pred));
}
